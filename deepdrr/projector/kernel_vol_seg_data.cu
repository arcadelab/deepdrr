/*
 * This file contains the declarations of the CUDA textures for:
 *  - NUM_VOLUMES CT volumes
 *  - (NUM_MATERIALS * NUM_VOLUMES) segmentation channels
 */

#define SEG_PASTER(vol_id, mat_id) seg_ ## vol_id ## _ ## mat_id
#define SEG(vol_id, mat_id) SEG_PASTER(vol_id, mat_id)
#define VOL_PASTER(vol_id) volume_ ## vol_id
#define VOLUME(vol_id) VOL_PASTER(vol_id)

#ifndef NUM_MATERIALS
#define NUM_MATERIALS 14
#endif

#ifndef NUM_VOLUMES
#define NUM_VOLUMES 1
#endif

#ifndef ATTENUATE_OUTSIDE_VOLUME
#define ATTENUATE_OUTSIDE_VOLUME 0
#endif

#ifndef AIR_DENSITY
#define AIR_DENSITY 0.1129 
#endif

#ifndef AIR_INDEX
#define AIR_INDEX 0
#endif

/*** Handle one volume ***/
#if NUM_VOLUMES > 0
#define CURR_VOL_ID 0
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle two volumes ***/
#if NUM_VOLUMES > 1
#define CURR_VOL_ID 1
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle three volumes ***/
#if NUM_VOLUMES > 2
#define CURR_VOL_ID 2
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle four volumes ***/
#if NUM_VOLUMES > 3
#define CURR_VOL_ID 3
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle five volumes ***/
#if NUM_VOLUMES > 4
#define CURR_VOL_ID 4
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle six volumes ***/
#if NUM_VOLUMES > 5
#define CURR_VOL_ID 5
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle seven volumes ***/
#if NUM_VOLUMES > 6
#define CURR_VOL_ID 6
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle eight volumes ***/
#if NUM_VOLUMES > 7
#define CURR_VOL_ID 7
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle nine volumes ***/
#if NUM_VOLUMES > 8
#define CURR_VOL_ID 8
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif


/*** Handle ten volumes ***/
#if NUM_VOLUMES > 9
#define CURR_VOL_ID 9
// the CT volume
texture<float, 3, hipReadModeElementType> VOLUME(CURR_VOL_ID);

// channel of the materials array, same size as the volume.
#if NUM_MATERIALS > 0
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 0);
#endif
#if NUM_MATERIALS > 1
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 1);
#endif
#if NUM_MATERIALS > 2
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 2);
#endif
#if NUM_MATERIALS > 3
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 3);
#endif
#if NUM_MATERIALS > 4
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 4);
#endif
#if NUM_MATERIALS > 5
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 5);
#endif
#if NUM_MATERIALS > 6
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 6);
#endif
#if NUM_MATERIALS > 7
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 7);
#endif
#if NUM_MATERIALS > 8
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 8);
#endif
#if NUM_MATERIALS > 9
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 9);
#endif
#if NUM_MATERIALS > 10
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 10);
#endif
#if NUM_MATERIALS > 11
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 11);
#endif
#if NUM_MATERIALS > 12
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 12);
#endif
#if NUM_MATERIALS > 13
texture<float, 3, hipReadModeElementType> SEG(CURR_VOL_ID, 13);
#endif

#undef CURR_VOL_ID
#endif
